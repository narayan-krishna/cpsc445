
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <math.h>

using namespace std;

//read string from file into a vector -> translate chars to ints
void read_csv(vector<float> &values, const string &path/*, int &new_line_loc*/){
    ifstream input_stream (path);

    if (!input_stream.is_open()) {
      cerr << "coudn't find/open file..." << endl;
      exit(EXIT_FAILURE);
    }

    for(string line; getline(input_stream, line);) {
      stringstream ss(line);

      string float_string;
      while(getline(ss, float_string, ',')) {
        values.push_back( (float)atof(float_string.c_str()) );
      }
    }
}

//print a sequence of characters to a file
void print_to_csv(const float *sequence, int length, string output_file) {

  ofstream out_file;
  out_file.open (output_file);

  for(int i = 0; i < length; i ++) {
    out_file << sequence[i] << endl;
  }

  out_file.close();
  // system("head output.csv");
  // system("rm output.csv");
}

__global__ 
void sqrt(float *da, int N) {

  int tid = threadIdx.x;
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float s[512];
  //allocated 512 floats per block
  //copy over 512 floats into the corresponding block

  s[tid] = da[gid];
  __syncthreads();
  da[gid] = sqrt(s[tid]);
  // printf("tid is: %i, seeing value: %f\n", tid, da[tid]);
}

int main() {
  cout << "\ncsv input head --------------------" << endl;
  system("head input.csv");
  cout << "\n-----------------------------" << endl;

  //INPUTS
  int N = 0;

  //read into vector so for dynamic length + size checking
  vector<float> inputs;
  read_csv(inputs, "input.csv");
  cout << "\ninputs(" << inputs.size() << ")" << endl;

  N = inputs.size();

  float *ha = new float[N];

  for(int i = 0; i < N; i++) {
    ha[i] = inputs[i];
  }

  for(int i = 0; i < 20; i ++) {
    cout << inputs[i] << ", ";
  }
  cout << "..." << endl; cout << endl;

  float *da;
  hipMalloc((void **) &da, N*sizeof(float));
  hipMemcpy(da, ha, N*sizeof(float), hipMemcpyHostToDevice); //copy ints from ha into da

  int Nthreads = 512;
  int Nblocks = (N + (Nthreads - 1)) / Nthreads;
  cout << Nthreads << ", " << Nblocks << endl;
  sqrt<<<Nblocks,Nthreads>>>(da, N);
  hipDeviceSynchronize();

  hipMemcpy(ha, da, N*sizeof(float), hipMemcpyDeviceToHost); //copy back value of da int sum

  print_to_csv(ha, N, "output.csv");

  cout << "head output csv" << "--------------" << endl;
  system("head output.csv");
  cout << "------------------------------------" << endl;

  hipFree(da);
  free(ha);
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <math.h>

using namespace std;

//read string from file into a vector -> translate chars to ints
void read_csv(vector<float> &values, const string &path, int &column_count){
    ifstream input_stream (path);

    if (!input_stream.is_open()) {
      cerr << "coudn't find/open file..." << endl;
      exit(EXIT_FAILURE);
    }

    //how many columns are there?

    for(string line; getline(input_stream, line); column_count ++) {
      cout << line << endl;
      stringstream ss(line);

      string float_string;
      while(getline(ss, float_string, ',')) {
        values.push_back( (float)atof(float_string.c_str()) );
      }
    }
}

inline void get_resident_coords(const int &index, int &x_coord, int &y_coord, int &rows) {
    x_coord = (index / rows);
    y_coord = (index % rows); 
}

//print a sequence of characters to a file
void print_to_csv(const bool *sequence, int columns, int rows, string output_file) {

  ofstream out_file;
  out_file.open (output_file);

  // for(int i = 0; i < length; i ++) {
  //   if (sequence[i] == 1) {
  //     int x_coord; int y_coord;
  //     get_resident_coords(i, x_coord, y_coord, rows);
  //     out_file << x_coord-1 << ", " << y_coord-1 << endl; 
  //   }
  // }
  for(int i = 0; i < rows; i ++) {
    for(int j = 0; j < columns; j ++) {
      if (sequence[(i*(columns))+j] == 1) {
        out_file << i << ", " << j;
        out_file << " (" << sequence[(i*columns)+j] << ")" << endl;
      }
    }
  }

  out_file.close();
  // system("head output.csv");
  // system("rm output.csv");
}

__device__
bool is_smaller_or_greater(float *da, const int &addr_1d, const int &rows, const int &N) {
  // cout << here << endl;
  // bool check_for_smaller = false;
  // bool decided = false;

  int neighbors[8]; //eight surrounding neighbors
  neighbors[0] = addr_1d - 1;
  neighbors[1] = addr_1d + 1;

  neighbors[2] = addr_1d - (rows);

  neighbors[3] = addr_1d - (rows) - 1;
  neighbors[4] = addr_1d - (rows) + 1;

  neighbors[5] = addr_1d + (rows);

  neighbors[6] = addr_1d + (rows) - 1;
  neighbors[7] = addr_1d + (rows) + 1;

  // for(int i = 0; i < 8; i ++) {
  //   if(neighbors[i] != 0) { //ignore if nieghbor is negative/outofgrid
  //     //is the nieghbor smaller than the current cell?
  //     bool is_smaller = (da[neighbors[i]] < da[addr_1d]);
  //     if (decided) { //if we already know we're looking for g/s
  //       if (is_smaller != check_for_smaller) { //if we dont' match the condition 
  //                                              //we're checking for
  //         return false; //return false
  //       }
  //     } else { //if we haven't decided, decided will be this 
  //       check_for_smaller = is_smaller;
  //       decided = true;
  //     }
  //   }
  // }
  // return true;
  // printf("%i [%f] -- %f,%f,%f,%f,%f,%f,%f,%f\n", addr_1d, da[addr_1d],
                                        // da[neighbors[0]],
                                        // da[neighbors[1]],
                                        // da[neighbors[2]],
                                        // da[neighbors[3]],
                                        // da[neighbors[4]],
                                        // da[neighbors[5]],
                                        // da[neighbors[6]],
                                        // da[neighbors[7]]);
  if(da[addr_1d] == 0) return false;
  bool greater = true; bool lesser = true;
  for(int i = 0; i < 8; i ++) {
    if (da[neighbors[i]] != 0) {
      if(da[neighbors[i]] < da[addr_1d]) {
        greater = false;
      } else {
        lesser = false;
      }
    }
  } 

  if(greater || lesser) {
    return true;
  } else {
    return false;
  }
}

__global__ 
void extreme(float *da, bool *dbools, int N, int rows, int columns) {

  // int tid = threadIdx.x;
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  // __shared__ float s[512];
  //allocated 512 floats per block
  //copy over 512 floats into the corresponding block

  // s[tid] = da[gid]; //copy everything
  // __syncthreads();
  dbools[gid] = is_smaller_or_greater(da, gid, rows, N);
  // printf("tid is: %i, seeing value: %f\n", tid, da[tid]);
}

/**
 * this program will
 * take an array of elements
 * it will check its neighbors
 * */

int main() {
  cout << "\ncsv input head --------------------" << endl;
  system("head input.csv");
  cout << "\n-----------------------------" << endl;

  //INPUTS
  int N = 0;
  int rows = 0; int columns = 0;

  //read into vector so for dynamic length + size checking
  vector<float> inputs;
  read_csv(inputs, "input.csv", columns);
  cout << "\ninputs(" << inputs.size() << ")" << endl;

  N = inputs.size();

  rows = N/columns;
  printf("rows: %i | columns: %i\n", rows, columns);

  vector<vector<float>> inputs_2d(rows + 2, vector<float> (columns + 2, 0));

  int o = 0;
  for(int i = 1; i < rows + 1; i ++) {
    for(int j = 1; j < columns + 1; j ++) {
      inputs_2d[i][j] = inputs[o];
      o++;
    }
  }

  for(int i = 0; i < rows + 2; i ++) {
    for(int j = 0; j < columns + 2; j ++) {
      cout << inputs_2d[i][j];
    }
    cout << endl;
  }


  int Ndeadcells = (rows + 2) * (columns + 2);
  float *ha = new float[Ndeadcells];
  bool *hbools = new bool[Ndeadcells]();

  int k = 0;
  for(int i = 0; i < rows + 2; i ++) {
    for(int j = 0; j < columns + 2; j ++) {
      // cout << inputs_2d[i][j];
      ha[k] = inputs_2d[i][j];
      k++;
    }
    // cout << endl;
  }

  cout << "flattened" << endl;
  for(int i = 0; i < Ndeadcells; i ++) {
    cout << ha[i] << "," ;
  } cout << endl;

  for(int i = 0; i < 20; i ++) {
    cout << inputs[i] << ", ";
  }
  cout << "..." << endl; cout << endl;

  float *da; bool *dbools;
  hipMalloc((void **) &da, Ndeadcells*sizeof(float));
  hipMalloc((void **) &dbools, Ndeadcells*sizeof(bool));

  hipMemcpy(da, ha, Ndeadcells*sizeof(float), hipMemcpyHostToDevice); //copy ints from ha into da
  hipMemcpy(dbools, hbools, Ndeadcells*sizeof(bool), hipMemcpyHostToDevice); //copy ints from ha into da

  int Nthreads = 512;
  int Nblocks = (Ndeadcells + (Nthreads - 1)) / Nthreads;
  cout << Nthreads << ", " << Nblocks << endl;

  extreme<<<Nblocks,Nthreads>>>(da, dbools, Ndeadcells, rows + 2, columns + 2);
  hipDeviceSynchronize();

  hipMemcpy(ha, da, Ndeadcells*sizeof(float), hipMemcpyDeviceToHost); //copy back value of da int sum
  hipMemcpy(hbools, dbools, Ndeadcells*sizeof(bool), hipMemcpyDeviceToHost); //copy back value of da int sum

  for(int i = 0; i < rows + 2; i ++) {
    for(int j = 0; j < columns + 2; j ++) {
      cout << hbools[(i*(columns + 2))+j];
    }
    cout << endl;
    // cout << endl;
  }

  print_to_csv(hbools, columns + 2, rows + 2, "output.csv");

  cout << "head output csv" << "--------------" << endl;
  system("cat output.csv");
  cout << "------------------------------------" << endl;

  hipFree(da);
  free(ha);
  return 0;
}